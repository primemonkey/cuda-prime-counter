#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

__global__ void primeCount(unsigned long int* numbers, int* primes, long size) 
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index < size) 
    {
        unsigned long int number = numbers[index];
        
        if (number < 2) 
            primes[index] = 0;
        
        else 
        {
            primes[index] = 1;
            for (unsigned long int i = 2; i * i <= number; i++) 
            {
                if (number % i == 0) 
                {
                    primes[index] = 0;
                    break;
                }
            }
        }
    }
}

int main(int argc,char **argv) 
{
    Args ins__args;
    parseArgs(&ins__args, &argc, argv);

    //program input argument
    long inputArgument = ins__args.arg; 
    unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
    numgen(inputArgument, numbers);

    struct timeval ins__tstart, ins__tstop;
    gettimeofday(&ins__tstart, NULL);

    // allocate memory
    unsigned long int *device_numbers;
    int *device_primes;
    
    hipMalloc(&device_numbers, inputArgument * sizeof(unsigned long int));
    hipMalloc(&device_primes, inputArgument * sizeof(int));

    // numbers to device
    hipMemcpy(device_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice);

    // run your CUDA kernel(s) here
    int blockSize = 256;
    int numBlocks = (inputArgument + blockSize - 1) / blockSize;
    primeCount<<<numBlocks, blockSize>>>(device_numbers, device_primes, inputArgument);
	
    /* 
    for(long i=0;i<inputArgument;i++)
    	printf("%ld\n",numbers[i]);
    */
    
    // primes to host
    int *primes = (int*)malloc(inputArgument * sizeof(int));
    hipMemcpy(primes, device_primes, inputArgument * sizeof(int), hipMemcpyDeviceToHost);

    // count
    int primeCount = 0;
    
    for (long i = 0; i < inputArgument; i++) 
        primeCount += primes[i];

    printf("Number of primes: %d\n", primeCount);

    // synchronize/finalize your CUDA computations
    hipDeviceSynchronize();

    gettimeofday(&ins__tstop, NULL);
    ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

    // free memory
    free(numbers);
    free(primes);
    hipFree(device_numbers);
    hipFree(device_primes);
}

